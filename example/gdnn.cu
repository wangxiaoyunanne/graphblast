#define GRB_USE_CUDA
#define private public

#include <iostream>
#include <algorithm>
#include <string>

#include <cstdio>
#include <cstdlib>

// #include <hip/hip_runtime_api.h>
#include <boost/program_options.hpp>

#include "graphblas/graphblas.hpp"
#include "graphblas/algorithm/dnn.hpp"
#include "graphblas/algorithm/common.hpp"
#include "test/test.hpp"

#define NMAXLAYERS 1 // 3
#define NMAXNEURONS 1 // 4

bool debug_;
bool memory_;

int main(int argc, char** argv) {
  
  // Parameters
  int maxLayers [NMAXLAYERS] = {120};
  int Nneurons [NMAXNEURONS] = {1024};
  double neuralNetBias [NMAXNEURONS] = {-0.3};
  // int maxLayers [NMAXLAYERS] = {120, 480, 1920};
  // int Nneurons [NMAXNEURONS] = {1024, 4096, 16384, 65536};
  // double neuralNetBias [NMAXNEURONS] = {-0.3, -0.35, -0.4, -0.45};
  int numFeatures = 60000;

  if (argc < 2) {
    fprintf(stderr, "Usage: %s [data_dir]\n", argv[0]);
    exit(1);
  }

  // Parse arguments
  bool debug;
  bool mtxinfo;
  bool filter;
  int  directed;
  char* dat_name;
  po::variables_map vm;

  // Parse args
  parseArgs(argc, argv, &vm);
  debug     = vm["debug"    ].as<bool>();
  mtxinfo   = vm["mtxinfo"  ].as<bool>();
  filter    = vm["filter"   ].as<bool>();
  directed  = vm["directed" ].as<int>();

  for (int i = 0; i < NMAXNEURONS; i++) {
    for (int j = 0; j < NMAXLAYERS; j++) {

      // Parameters for this iteration
      int numNeurons = Nneurons[i];
      float bias = neuralNetBias[i];
      int nlayers = maxLayers[j];

      // File names
      std::string true_categories_file_path = std::string(argv[argc-1]) + "/DNN/neuron" + std::to_string(numNeurons) + "-l" + std::to_string(nlayers) + "-categories.tsv";
      std::string input_features_file_path = std::string(argv[argc-1]) + "/MNIST/sparse-images-" + std::to_string(numNeurons) + ".mtx";
      std::string layers_file_prefix = std::string(argv[argc-1]) + "/DNN/neuron" + std::to_string(numNeurons) + "/n" + std::to_string(numNeurons) + "-l";

      std::vector<graphblas::Index> row_indices, row_idx_mnist, bias_idx(numNeurons);
      std::vector<graphblas::Index> col_indices, col_idx_mnist;
      std::vector<float> values, val_mnist, bias_v(numNeurons, bias);
      std::vector<int> true_categories_idx;
      std::vector<bool> true_categories(numFeatures, 0);
      graphblas::Index nrows, ncols, nvals, nrow_mnist, ncol_mnist, nval_mnist;

      // // Vectors to build bias MATRIX
      // std::vector<graphblas::Index> row_idx_bias(numNeurons);
      // std::iota(std::begin(row_idx_bias), std::end(row_idx_bias), 0);
      // std::vector<graphblas::Index> col_idx_bias(numNeurons);
      // std::iota(std::begin(col_idx_bias), std::end(col_idx_bias), 0);
      // std::vector<float> diag_val_bias(numNeurons);
      // std::fill(std::begin(diag_val_bias), std::end(diag_val_bias), bias);

      std::vector<graphblas::Matrix<float>> Weights(nlayers, graphblas::Matrix<float>(numNeurons, numNeurons));
      // std::vector<graphblas::Matrix<float>> Biases(nlayers, graphblas::Matrix<float>(numNeurons, numNeurons));
      graphblas::Vector<bool> TrueCategories(numFeatures);

      // Read true categories
      std::ifstream categories_file;
      categories_file.open(true_categories_file_path.c_str());
      int x;
      while (categories_file >> x) {
        true_categories_idx.push_back(x);
        true_categories[x-1] = 1;
      }
      TrueCategories.build(&true_categories, numFeatures);
      if (debug)
        CHECK(TrueCategories.print());

      // Read input features
      readMtx(input_features_file_path.c_str(), &row_idx_mnist, &col_idx_mnist, &val_mnist, &nrow_mnist, &ncol_mnist, &nval_mnist, directed, mtxinfo, NULL);
      std::cout << input_features_file_path << std::endl;
      std::cout << nrow_mnist << ", " << ncol_mnist << std::endl;
      graphblas::Matrix<float> mnist(nrow_mnist, ncol_mnist);
      CHECK(mnist.build(&row_idx_mnist, &col_idx_mnist, &val_mnist, nval_mnist, GrB_NULL, NULL));
      CHECK(mnist.nrows(&nrows));
      CHECK(mnist.ncols(&ncols));
      CHECK(mnist.nvals(&nvals));
      if (debug)
        CHECK(mnist.print());
      
      // Read weights
      for (int layer = 0; layer < nlayers; layer++)
      {
          // Read mtx file of layers
          std::string file_name = layers_file_prefix + std::to_string(layer+1) + ".mtx";
          readMtx(file_name.c_str() , &row_indices, &col_indices, &values, &nrows, &ncols, &nvals, directed, mtxinfo, NULL);
          std::cout<< file_name << std::endl;
          
          // Build matrix
          CHECK((Weights[layer]).build(&row_indices, &col_indices, &values, nvals, GrB_NULL, dat_name));
          // CHECK(Weights[layer].nrows(&nrows));
          // CHECK(Weights[layer].ncols(&ncols));
          // CHECK(Weights[layer].nvals(&nvals));
          if (debug)
            CHECK(Weights[layer].print());
          // bias MATRIX
          // graphblas::Matrix<float> b(nrows, ncols);
          // CHECK(b.build(&row_idx_b, &col_idx_b, &diag_val_b, numNeurons, GrB_NULL, dat_name));
          // Biases[layer] = b;
          // CHECK(b.print());
      }

      // bias VECTOR
      // graphblas ::  Vector<float> Biases(nrows);
      // for (int i =0; i < numNeurons; i++) {bias_idx [i] = i ;}
      // CHECK(Biases.build(&bias_idx, &bias_v, nrows,GrB_NULL));
      Vector<float> Biases(nrows);
      CHECK(Biases.fill(bias));

      /*!
       * This is an imperfect solution, because this should happen in
       * desc.loadArgs(vm) instead of application code!
       * TODO(@ctcyang): fix this
       */
      graphblas::Descriptor desc;
      CHECK(desc.loadArgs(vm));

      // // Cpu BFS
      // CpuTimer dnn_cpu;
      // float* h_dnn_cpu = reinterpret_cast<float*>(malloc(nrows*sizeof(float)));
      // int depth = 10000;
      // dnn_cpu.Start();
      // int d = graphblas::algorithm::dnnCpu(...);
      // dnn_cpu.Stop();

      // Warmup
      CpuTimer warmup;
      warmup.Start();
      graphblas::algorithm::dnn(numNeurons, numFeatures, 
                                mnist, Weights, Biases, 
                                true, /*TrueCategories,*/ true_categories, // Alternative: dense vector
                                filter, &desc);
      warmup.Stop();

      // // Benchmark
      // CpuTimer dnn_gpu_timer;
      // // hipProfilerStart();
      // dnn_gpu_timer.Start();
      // for (int i = 0; i < niter; i++) {
      //   graphblas::algorithm::dnn(...);
      // }
      // // hipProfilerStop();
      // dnn_gpu_timer.Stop();

      // float flop = 0;
      // std::cout << "cpu, " << dnn_cpu.ElapsedMillis() << ", \n";
      // std::cout << "warmup, " << warmup.ElapsedMillis() << ", " <<
      //   flop/warmup.ElapsedMillis()/1000000.0 << "\n";
      // float elapsed_dnn_gpu = dnn_gpu_timer.ElapsedMillis();

      // // if (niter) {
      // //   std::vector<float> h_dnn_gpu2;
      // //   CHECK(y.extractTuples(&h_dnn_gpu2, &nrows));
      // //   BOOST_ASSERT_LIST_FLOAT(h_dnn_cpu, h_dnn_gpu2, nrows);
      // // }

    }
  }
  

  return 0;
}
